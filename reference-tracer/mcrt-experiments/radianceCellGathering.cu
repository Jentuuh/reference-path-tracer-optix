#include "hip/hip_runtime.h"
#pragma once

#include <optix_device.h>
#include "random.hpp"
#include "vec_math.hpp"

#include "LaunchParams.hpp"
#include "glm/glm.hpp"

#include "spherical_harmonics.cuh"

#define PI 3.14159265358979323846f
#define EPSILON 0.0000000000002f
#define NUM_SAMPLES_PER_STRATIFY_CELL 10

using namespace mcrt;

namespace mcrt {

    extern "C" __constant__ LaunchParamsRadianceCellGather optixLaunchParams;

    static __forceinline__ __device__
        void* unpackPointer(uint32_t i0, uint32_t i1)
    {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void* ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    }

    static __forceinline__ __device__
        void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    static __forceinline__ __device__ RadianceCellGatherPRD loadRadianceCellGatherPRD()
    {
        RadianceCellGatherPRD prd = {};

        prd.distanceToClosestProxyIntersectionSquared = __uint_as_float(optixGetPayload_0());
        prd.rayOrigin.x = __uint_as_float(optixGetPayload_1());
        prd.rayOrigin.y = __uint_as_float(optixGetPayload_2());
        prd.rayOrigin.z = __uint_as_float(optixGetPayload_3());

        return prd;
    }

    static __forceinline__ __device__ void storeRadianceCellGatherPRD(RadianceCellGatherPRD prd)
    {
        optixSetPayload_0(__float_as_uint(prd.distanceToClosestProxyIntersectionSquared));
        optixSetPayload_1(__float_as_uint(prd.rayOrigin.x));
        optixSetPayload_2(__float_as_uint(prd.rayOrigin.y));
        optixSetPayload_3(__float_as_uint(prd.rayOrigin.z));
    }


    extern "C" __global__ void __closesthit__radiance__cell__gathering__scene()
    {
        const MeshSBTDataRadianceCellGather& sbtData
            = *(const MeshSBTDataRadianceCellGather*)optixGetSbtDataPointer();

        const int   primID = optixGetPrimitiveIndex();
        const glm::ivec3 index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        const glm::vec3 intersectionWorldPos =
            (1.f - u - v) * sbtData.vertex[index.x]
            + u * sbtData.vertex[index.y]
            + v * sbtData.vertex[index.z];

        RadianceCellGatherPRD prd = loadRadianceCellGatherPRD();
        float distanceToProxyIntersect = (((intersectionWorldPos.x - prd.rayOrigin.x) * (intersectionWorldPos.x - prd.rayOrigin.x)) + ((intersectionWorldPos.y - prd.rayOrigin.y) * (intersectionWorldPos.y - prd.rayOrigin.y)) + ((intersectionWorldPos.z - prd.rayOrigin.z) * (intersectionWorldPos.z - prd.rayOrigin.z)));

        prd.distanceToClosestProxyIntersectionSquared = distanceToProxyIntersect;
        storeRadianceCellGatherPRD(prd);
    }

    extern "C" __global__ void __anyhit__radiance__cell__gathering__scene() {
        // Do nothing
        printf("Hit scene!");
    }

    extern "C" __global__ void __miss__radiance__cell__gathering()
    {
    }

    extern "C" __global__ void __raygen__renderFrame__cell__gathering()
    {
        // Get thread indices
        const int uIndex = optixGetLaunchIndex().x;
        const int vIndex = optixGetLaunchIndex().y;

        // Amount SH basis functions
        int amountBasisFunctions = optixLaunchParams.sphericalHarmonicsWeights.amountBasisFunctions;

        // Size of a radiance cell + dimensions of each stratified cell
        const float cellSize = optixLaunchParams.cellSize;
        float stratifyCellWidth = cellSize / optixLaunchParams.stratifyResX;
        float stratifyCellHeight = cellSize / optixLaunchParams.stratifyResY;

        float stratifyCellWidthNormalized = 1.0 / optixLaunchParams.stratifyResX;
        float stratifyCellHeightNormalized = 1.0 / optixLaunchParams.stratifyResY;

        // TODO: SKIP PIXELS THAT ARE BLACK!
        uint32_t lightSrcColor = optixLaunchParams.lightSourceTexture.colorBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex];
        //printf("%d", lightSrcColor);

        glm::vec3 UVWorldPos = optixLaunchParams.uvWorldPositions.UVDataBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex].worldPosition;
        const glm::vec3 UVNormal = optixLaunchParams.uvWorldPositions.UVDataBuffer[vIndex * optixLaunchParams.lightSourceTexture.size + uIndex].worldNormal;
        float3 uvNormal3f = float3{ UVNormal.x, UVNormal.y, UVNormal.z };

        // We apply a small offset of 0.00001f in the direction of the normal to the UV world pos, to 'mitigate' floating point rounding errors causing false occlusions/illuminations
        UVWorldPos = glm::vec3{ UVWorldPos.x + UVNormal.x * 0.00001f, UVWorldPos.y + UVNormal.y * 0.00001f, UVWorldPos.z + UVNormal.z * 0.00001f };
        
        // Iterate over all non-empty cells
        for (int i = 0; i < optixLaunchParams.nonEmptyCells.size; i++)
        {
            // Take different seed for each radiance cell
            unsigned int seed = tea<4>(vIndex * optixLaunchParams.lightSourceTexture.size + uIndex, i);

            glm::vec3 cellCenter = optixLaunchParams.nonEmptyCells.centers[i];
            glm::vec3 lightToCellDir = { cellCenter.x - UVWorldPos.x, cellCenter.y - UVWorldPos.y, cellCenter.z - UVWorldPos.z };

            float3 rayOrigin3f = float3{ UVWorldPos.x, UVWorldPos.y, UVWorldPos.z };
            float3 rayOgToCellCenter3f = float3{ lightToCellDir.x, lightToCellDir.y, lightToCellDir.z };
            
            // Cosine between vector from ray origin to cell center and texel normal to check if cell is facing
            double radCellFacing = dot(normalize(rayOgToCellCenter3f), uvNormal3f);

            if (radCellFacing > 0)
            {
                float3 ogLeft{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };
                float3 ogRight{ cellCenter.x + 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
                float3 ogUp{ cellCenter.x - 0.5f * cellSize, cellCenter.y + 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
                float3 ogDown{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };
                float3 ogFront{ cellCenter.x - 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z - 0.5f * cellSize };
                float3 ogBack{ cellCenter.x + 0.5f * cellSize, cellCenter.y - 0.5f * cellSize, cellCenter.z + 0.5f * cellSize };


                // LEFT, RIGHT, UP, DOWN, FRONT, BACK
                float3 cellNormals[6] = { float3{-1.0f, 0.0f, 0.0f}, float3{1.0f, 0.0f, 0.0f}, float3{0.0f, 1.0f, 0.0f}, float3{0.0f, -1.0f, 0.0f}, float3{0.0f, 0.0f, -1.0f}, float3{0.0f, 0.0f, 1.0f} };
                // Origin, du, dv for each face
                float3 faceOgDuDv[6][3] = { {ogLeft, float3{0.0f, 0.0f, -1.0f}, float3{0.0f, 1.0f, 0.0f} }, {ogRight, float3{0.0f, 0.0f, 1.0f},float3{0.0f, 1.0f, 0.0f} }, {ogUp, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 0.0f, 1.0f} }, {ogDown, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 0.0f, -1.0f}}, {ogFront, float3{1.0f, 0.0f, 0.0f},float3{0.0f, 1.0f, 0.0f} }, {ogBack, float3{-1.0f, 0.0f, 0.0f},float3{0.0f, 1.0f, 0.0f} } };
                // The indices of the SHs that belong to each face, to use while indexing the buffer (L,R,U,D,F,B)
                int4 cellSHIndices[6] = { int4{4, 0, 6, 2}, int4{1, 5, 3, 7}, int4{2, 3, 6, 7}, int4{4, 5, 0, 1}, int4{0, 1, 2, 3}, int4{5, 4, 7, 6} };

            
                for (int face = 0; face < 6; face++)
                {
                    // We accumulate the projections of each ray into this buffer, the projection of the lighting function
                    // onto the SH basis functions is estimated by Monte Carlo integration. 
                    double contribution[9] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
                    // Expected value buffer for weightA, weightB, weightC, weightD for bilinear interpolation in the face square/
                    // The expected values of these weights will be used to decide the final contribution to the SH of each corner after projection.
                    double bilinInterpolWeightsExpectedValues[4] = {0.0f, 0.0f, 0.0f, 0.0f};
                    // Rays that pass all tests and thus contribute
                    int n_samples = 0;

                    // Check if the current cell face is facing, otherwise skip.
                    double cellFaceFacing = dot(uvNormal3f, cellNormals[face]);
                    if (cellFaceFacing < 0)
                    {
                        // For each stratified cell on the face, take samples
                        for (int stratifyIndexX = 0; stratifyIndexX < optixLaunchParams.stratifyResX; stratifyIndexX++)
                        {
                            for (int stratifyIndexY = 0; stratifyIndexY < optixLaunchParams.stratifyResY; stratifyIndexY++)
                            {
                                glm::vec3 og = glm::vec3{ faceOgDuDv[face][0].x,faceOgDuDv[face][0].y,faceOgDuDv[face][0].z };
                                glm::vec3 du = glm::vec3{ faceOgDuDv[face][1].x,faceOgDuDv[face][1].y,faceOgDuDv[face][1].z };
                                glm::vec3 dv = glm::vec3{ faceOgDuDv[face][2].x,faceOgDuDv[face][2].y,faceOgDuDv[face][2].z };

                                glm::vec3 stratifyCellOrigin = og + (stratifyIndexX * stratifyCellWidth * du) + (stratifyIndexY * stratifyCellHeight * dv);

                                // Send out a ray for each sample
                                for (int sample = 0; sample < NUM_SAMPLES_PER_STRATIFY_CELL; sample++)
                                {
                                    // Take a random sample on the face's stratified cell
                                    float2 randomOffset = float2{ rnd(seed), rnd(seed) };        
                                    glm::vec3 rayDestination = stratifyCellOrigin + (randomOffset.x * stratifyCellWidth * du) + (randomOffset.y * stratifyCellHeight * dv);

                                    // Ray direction
                                    glm::vec3 rayDir = rayDestination - UVWorldPos;

                                    // Convert to float3 format
                                    float3 rayOrigin3f = float3{ UVWorldPos.x, UVWorldPos.y, UVWorldPos.z };
                                    float3 rayDir3f = float3{ rayDir.x, rayDir.y, rayDir.z };

                                    // Calculate spherical coordinate representation of ray
                                    // (https://en.wikipedia.org/wiki/Spherical_coordinate_system#Cartesian_coordinates)
                                    float3 normalizedRayDir = normalize(rayDir3f);
                                    double theta = acos(normalizedRayDir.z);
                                    int signY = signbit(normalizedRayDir.y) == 0 ? 1 : -1;
                                    double phi = signY * acos(normalizedRayDir.x / (sqrtf((normalizedRayDir.x * normalizedRayDir.x) + (normalizedRayDir.y * normalizedRayDir.y))));

                                    printf("%f %f", theta, phi);

                                    RadianceCellGatherPRD prd{};
                                    prd.rayOrigin = UVWorldPos;

                                    unsigned int u0, u1, u2, u3;

                                    u1 = __float_as_uint(prd.rayOrigin.x);
                                    u2 = __float_as_uint(prd.rayOrigin.y);
                                    u3 = __float_as_uint(prd.rayOrigin.z);

                                    // Call against scene geometry
                                    optixTrace(optixLaunchParams.sceneTraversable,
                                        rayOrigin3f,
                                        rayDir3f,
                                        0.f,    // tmin
                                        1e20f,  // tmax
                                        0.0f,   // rayTime
                                        OptixVisibilityMask(255),
                                        OPTIX_RAY_FLAG_DISABLE_ANYHIT,      // We only need closest-hit for scene geometry
                                        0,  // SBT offset
                                        1,  // SBT stride
                                        0,  // missSBTIndex
                                        u0, u1, u2, u3
                                    );

                                    prd.distanceToClosestProxyIntersectionSquared = u0;
                                    float distanceToGridIntersect = (((rayDestination.x - prd.rayOrigin.x) * (rayDestination.x - prd.rayOrigin.x)) + ((rayDestination.y - prd.rayOrigin.y) * (rayDestination.y - prd.rayOrigin.y)) + ((rayDestination.z - prd.rayOrigin.z) * (rayDestination.z - prd.rayOrigin.z)));

                                    if (distanceToGridIntersect < prd.distanceToClosestProxyIntersectionSquared)
                                    {
                                        ++n_samples;

                                        // We calculate the dx and dy offsets to the (x,y) coordinate of the sampled point on a normalized square to use in 
                                        // the calculation of the weights for bilinear extrapolation
                                        float dx = (stratifyIndexX * stratifyCellWidthNormalized + randomOffset.x * stratifyCellWidthNormalized) - 0.5;
                                        float dy = (stratifyIndexY * stratifyCellHeightNormalized + randomOffset.y * stratifyCellHeightNormalized) - 0.5;
                       
                                        // Accumulate bilinear interpolation weights, see thesis for explanation
                                        bilinInterpolWeightsExpectedValues[0] += (0.5f + dx) * (1.0f - (0.5f + dy));
                                        bilinInterpolWeightsExpectedValues[1] += (1.0f - (0.5f + dx)) * (1.0f - (0.5f + dy));
                                        bilinInterpolWeightsExpectedValues[2] += (0.5f + dx) * (0.5f + dy);
                                        bilinInterpolWeightsExpectedValues[3] += (1.0f - (0.5f + dx)) * (0.5f + dy);

                                        // Project lighting function (single ray accumulation) onto SH basis functions
                                        contribution[0] += lightSrcColor * Y_0_0();
                                        contribution[1] += lightSrcColor * Y_min1_1(phi, theta);
                                        contribution[2] += lightSrcColor * Y_0_1(phi, theta);
                                        contribution[3] += lightSrcColor * Y_1_1(phi, theta);
                                        contribution[4] += lightSrcColor * Y_min2_2(phi, theta);
                                        contribution[5] += lightSrcColor * Y_min1_2(phi, theta);
                                        contribution[6] += lightSrcColor * Y_0_2(phi, theta);
                                        contribution[7] += lightSrcColor * Y_1_2(phi, theta);
                                        contribution[8] += lightSrcColor * Y_2_2(phi, theta);
                                    }
                                }
                            }
                        }
                    }

                    if (n_samples > 0)
                    {
                        // Divide by amount of rays that contributed (samples) to get expected weights value
                        double weightsFactor = 1.0 / n_samples;
                        for (int w = 0; w < 4; w++)
                        {
                            bilinInterpolWeightsExpectedValues[w] = bilinInterpolWeightsExpectedValues[w] * weightsFactor;
                        }

                        // Idem for SH basis coefficients, part of the Monte Carlo integration (see paper SH Lighting: 'The gritty details')
                        double weight = 1.0; // TODO: Unsure what this weight needs to be, in the paper they use 4pi because they are uniformly sampling the sphere, but that is not the case here...
                        double contributionFactor = weight / n_samples;
                        for (int w = 0; w < 9; w++)
                        {
                            contribution[w] = contribution[w] * contributionFactor;
                        }

                        // Current non-empty cell * amount of basis functions * 8 SHs per cell 
                        int cellOffset = i * amountBasisFunctions * 8;

                        double weightA = 1.0 / bilinInterpolWeightsExpectedValues[0];
                        double weightB = 1.0 / bilinInterpolWeightsExpectedValues[1];
                        double weightC = 1.0 / bilinInterpolWeightsExpectedValues[2];
                        double weightD = 1.0 / bilinInterpolWeightsExpectedValues[3];

                        for (int w = 0; w < 9; w++)
                        {                                                                                                   // Am i allowed to just add this here, won't this blow up the coefficients?
                            optixLaunchParams.sphericalHarmonicsWeights.weights[cellOffset + cellSHIndices[face].x * amountBasisFunctions + w] += contribution[w] * weightC;
                            optixLaunchParams.sphericalHarmonicsWeights.weights[cellOffset + cellSHIndices[face].y * amountBasisFunctions + w] += contribution[w] * weightD;
                            optixLaunchParams.sphericalHarmonicsWeights.weights[cellOffset + cellSHIndices[face].z * amountBasisFunctions + w] += contribution[w] * weightA;
                            optixLaunchParams.sphericalHarmonicsWeights.weights[cellOffset + cellSHIndices[face].w * amountBasisFunctions + w] += contribution[w] * weightB;
                        }
                    }
                }
            }
        }
    }
}