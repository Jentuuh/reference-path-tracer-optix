#include "hip/hip_runtime.h"
#pragma once

#include <optix_device.h>
#include "random.hpp"
#include "vec_math.hpp"

#include "LaunchParams.hpp"
#include "glm/glm.hpp"

#include "cube_mapping.cuh"
#include "utils.cuh"

#define PI 3.14159265358979323846f
#define EPSILON 0.0000000000002f
#define NUM_DIRECTION_SAMPLES 80
#define STRATIFY_X_RES 5
#define STRATIFY_Y_RES 5
#define PI_OVER_4 0.785398163397425f
#define PI_OVER_2 1.5707963267945f

using namespace mcrt;

namespace mcrt {
    extern "C" __constant__ LaunchParamsRadianceCellScatterUnbiased optixLaunchParams;

    static __forceinline__ __device__ RadianceCellScatterPRD loadRadianceCellScatterPRD()
    {
        RadianceCellScatterPRD prd = {};

        prd.distanceToClosestIntersectionSquared = __uint_as_float(optixGetPayload_0());
        prd.rayOrigin.x = __uint_as_float(optixGetPayload_1());
        prd.rayOrigin.y = __uint_as_float(optixGetPayload_2());
        prd.rayOrigin.z = __uint_as_float(optixGetPayload_3());

        return prd;
    }

    static __forceinline__ __device__ void storeRadianceCellScatterPRD(RadianceCellScatterPRD prd)
    {
        optixSetPayload_0(__float_as_uint(prd.distanceToClosestIntersectionSquared));
        optixSetPayload_1(__float_as_uint(prd.rayOrigin.x));
        optixSetPayload_2(__float_as_uint(prd.rayOrigin.y));
        optixSetPayload_3(__float_as_uint(prd.rayOrigin.z));
    }


    extern "C" __global__ void __closesthit__radiance__cell__scattering__scene__unbiased()
    {
        const MeshSBTDataRadianceCellScatter& sbtData
            = *(const MeshSBTDataRadianceCellScatter*)optixGetSbtDataPointer();

        const int primID = optixGetPrimitiveIndex();
        const glm::ivec3 index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        const glm::vec3 intersectionWorldPos =
            (1.f - u - v) * sbtData.vertex[index.x]
            + u * sbtData.vertex[index.y]
            + v * sbtData.vertex[index.z];

        RadianceCellScatterPRD prd = loadRadianceCellScatterPRD();
        float distanceToIntersectionSquared = (((intersectionWorldPos.x - prd.rayOrigin.x) * (intersectionWorldPos.x - prd.rayOrigin.x)) + ((intersectionWorldPos.y - prd.rayOrigin.y) * (intersectionWorldPos.y - prd.rayOrigin.y)) + ((intersectionWorldPos.z - prd.rayOrigin.z) * (intersectionWorldPos.z - prd.rayOrigin.z)));

        prd.distanceToClosestIntersectionSquared = distanceToIntersectionSquared;
        storeRadianceCellScatterPRD(prd);
    }

    extern "C" __global__ void __anyhit__radiance__cell__scattering__scene__unbiased() {
        // Do nothing
    }

    extern "C" __global__ void __miss__radiance__cell__scattering__unbiased()
    {
        // Do nothing
    }

    extern "C" __global__ void __raygen__renderFrame__cell__scattering__unbiased()
    {
        const int uvIndex = optixGetLaunchIndex().x;

        const int nonEmptyCellIndex = optixLaunchParams.nonEmptyCellIndex;

        // Take different seed for each radiance cell face
        unsigned int seed = tea<4>(uvIndex, nonEmptyCellIndex);

        // Get UV world position for this shader pass
        const int uvInsideOffset = optixLaunchParams.uvsInsideOffsets[nonEmptyCellIndex];
        glm::vec2 uv = optixLaunchParams.uvsInside[uvInsideOffset + uvIndex];
        const int u = int(uv.x * optixLaunchParams.uvWorldPositions.size);
        const int v = int(uv.y * optixLaunchParams.uvWorldPositions.size);

        glm::vec3 UVWorldPos = optixLaunchParams.uvWorldPositions.UVDataBuffer[v * optixLaunchParams.uvWorldPositions.size + u].worldPosition;
        const glm::vec3 UVNormal = optixLaunchParams.uvWorldPositions.UVDataBuffer[v * optixLaunchParams.uvWorldPositions.size + u].worldNormal;
        const glm::vec3 diffuseColor = optixLaunchParams.uvWorldPositions.UVDataBuffer[v * optixLaunchParams.uvWorldPositions.size + u].diffuseColor;

        // We apply a small offset of 0.00001f in the direction of the normal to the UV world pos, to 'mitigate' floating point rounding errors causing false occlusions/illuminations
        UVWorldPos = glm::vec3{ UVWorldPos.x + UVNormal.x * 0.0000001f, UVWorldPos.y + UVNormal.y * 0.0000001f, UVWorldPos.z + UVNormal.z * 0.0000001f };

        float3 rayOrigin3f = float3{ UVWorldPos.x, UVWorldPos.y, UVWorldPos.z };
        float3 uvNormal3f = float3{ UVNormal.x, UVNormal.y, UVNormal.z };

        // ======================================
        // Radiance + num of samples accumulators
        // ======================================
        glm::vec3 totalRadiance = glm::vec3{ 0.0f, 0.0f, 0.0f };
        int numSamples = 0;

        // ====================================================
        // Stratified sampling of indirect light source texture
        // ====================================================
        float stratifyWidth = float(optixLaunchParams.prevBounceTexture.size) / float(STRATIFY_X_RES);
        float stratifyHeight = float(optixLaunchParams.prevBounceTexture.size) / float(STRATIFY_Y_RES);
        for (int x = 0; x < STRATIFY_X_RES; x++)
        {
            int currentU = int(x * stratifyWidth);
            for (int y = 0; y < STRATIFY_Y_RES; y++)
            {
                int currentV = int(y * stratifyHeight);
                for (int i = 0; i < NUM_DIRECTION_SAMPLES; i++)
                {
                    // Generate random UVs in stratification cell
                    float2 uniformRandoms = float2{ rnd(seed), rnd(seed) };
                    glm::ivec2 uvLightSrc = glm::ivec2{ int(currentU + uniformRandoms.x * stratifyWidth), int(currentV + uniformRandoms.y * stratifyHeight) };
                    glm::vec3 lightSrcWorldPos = optixLaunchParams.uvWorldPositions.UVDataBuffer[uvLightSrc.y * optixLaunchParams.uvWorldPositions.size + uvLightSrc.x].worldPosition;
                    const glm::vec3 lightSrcNormal = optixLaunchParams.uvWorldPositions.UVDataBuffer[uvLightSrc.y * optixLaunchParams.uvWorldPositions.size + uvLightSrc.x].worldNormal;
                    lightSrcWorldPos += glm::vec3{lightSrcWorldPos.x + lightSrcNormal.x * 0.0001f, lightSrcWorldPos.y + lightSrcNormal.y * 0.0001f, lightSrcWorldPos.z + lightSrcNormal.z * 0.0001f };

                    uint32_t lightSrcColor = optixLaunchParams.prevBounceTexture.colorBuffer[uvLightSrc.y * optixLaunchParams.prevBounceTexture.size + uvLightSrc.x];

                    // Extract rgb values from indirect light source texel
                    uint32_t rLightSrc = 0x000000ff & (lightSrcColor);
                    uint32_t gLightSrc = (0x0000ff00 & (lightSrcColor)) >> 8;
                    uint32_t bLightSrc = (0x00ff0000 & (lightSrcColor)) >> 16;

                    glm::vec3 rayDir = lightSrcWorldPos - UVWorldPos;
                    float3 rayDir3f = float3{ rayDir.x, rayDir.y, rayDir.z };

                    // Prune any directions that barely have a contribution (to avoid excessive noise)
                    if (dot(normalize(rayDir3f), normalize(uvNormal3f)) < 0.4f)
                    {
                        continue;
                    }


                    RadianceCellScatterPRD prd;
                    unsigned int u0, u1, u2, u3;
                    u1 = __float_as_uint(prd.rayOrigin.x);
                    u2 = __float_as_uint(prd.rayOrigin.y);
                    u3 = __float_as_uint(prd.rayOrigin.z);

                    // Trace ray against scene geometry to see if ray is occluded
                    optixTrace(optixLaunchParams.sceneTraversable,
                        rayOrigin3f,
                        rayDir3f,
                        0.f,    // tmin
                        1e20f,  // tmax
                        0.0f,   // rayTime
                        OptixVisibilityMask(255),
                        OPTIX_RAY_FLAG_DISABLE_ANYHIT,      // We only need closest-hit for scene geometry
                        0,  // SBT offset
                        1,  // SBT stride
                        0,  // missSBTIndex
                        u0, u1, u2, u3
                    );

                    prd.distanceToClosestIntersectionSquared = __uint_as_float(u0);
                    float distanceToUVSquared = (((UVWorldPos.x - lightSrcWorldPos.x) * (UVWorldPos.x - lightSrcWorldPos.x)) + ((UVWorldPos.y - lightSrcWorldPos.y) * (UVWorldPos.y - lightSrcWorldPos.y)) + ((UVWorldPos.z - lightSrcWorldPos.z) * (UVWorldPos.z - lightSrcWorldPos.z)));

                    // If the ray is not occluded, we can let it contribute
                    if (distanceToUVSquared < prd.distanceToClosestIntersectionSquared)
                    {
                        // Cosine weighted contribution
                        float cosContribution = dot(normalize(rayDir3f), normalize(uvNormal3f));
                        totalRadiance += glm::vec3{ cosContribution * rLightSrc, cosContribution * gLightSrc, cosContribution * bLightSrc };
                        ++numSamples;
                    }
                }
            }
        }


        // TODO: add Monte Carlo weight (although how do we get the exact pdf that we are sampling from...?)
        const int r_result = int((totalRadiance.x / (float(numSamples))));
        const int g_result = int((totalRadiance.y / (float(numSamples))));
        const int b_result = int((totalRadiance.z / (float(numSamples))));

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000
            | (r_result << 0) | (g_result << 8) | (b_result << 16);

        optixLaunchParams.currentBounceTexture.colorBuffer[v * optixLaunchParams.currentBounceTexture.size + u] = rgba;
    }
}